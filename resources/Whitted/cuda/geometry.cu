#include "hip/hip_runtime.h"
//
// Copyright (c) 2019, NVIDIA CORPORATION. All rights reserved.
//
// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions
// are met:
//  * Redistributions of source code must retain the above copyright
//    notice, this list of conditions and the following disclaimer.
//  * Redistributions in binary form must reproduce the above copyright
//    notice, this list of conditions and the following disclaimer in the
//    documentation and/or other materials provided with the distribution.
//  * Neither the name of NVIDIA CORPORATION nor the names of its
//    contributors may be used to endorse or promote products derived
//    from this software without specific prior written permission.
//
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
// EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
// IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
// PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
// CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
// EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
// PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
// PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
// OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
// (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
// OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
//

#include <optix.h>

#include "optixWhitted.h"
#include "helpers.h"

extern "C" {
__constant__ Params params;
}

extern "C" __global__ void __intersection__parallelogram()
{
    const Parallelogram* floor = reinterpret_cast<Parallelogram*>( optixGetSbtDataPointer() );

    const float3 ray_orig = optixGetWorldRayOrigin();
    const float3 ray_dir  = optixGetWorldRayDirection();
    const float ray_tmin = optixGetRayTmin(), ray_tmax = optixGetRayTmax();

    float3 n = make_float3( floor->plane );
    float dt = dot(ray_dir, n );
    float t = (floor->plane.w - dot(n, ray_orig))/dt;
    if( t > ray_tmin && t < ray_tmax )
    {
        float3 p = ray_orig + ray_dir * t;
        float3 vi = p - floor->anchor;
        float a1 = dot(floor->v1, vi);
        if(a1 >= 0 && a1 <= 1)
        {
            float a2 = dot(floor->v2, vi);
            if(a2 >= 0 && a2 <= 1)
            {
                optixReportIntersection(
                    t,
                    0,
                    float3_as_args(n),
                    float_as_int( a1 ), float_as_int( a2 )
                    );
            }
        }
    }
}


extern "C" __device__ void intersect_sphere()
{
    const bool use_robust_method = true;

    const Sphere* sphere   = reinterpret_cast<Sphere*>( optixGetSbtDataPointer() );
    const float3  ray_orig = optixGetWorldRayOrigin();
    const float3  ray_dir  = optixGetWorldRayDirection();
    const float   ray_tmin = optixGetRayTmin(), ray_tmax = optixGetRayTmax();

    float3 O = ray_orig - sphere->center;
    float  l = 1 / length(ray_dir);
    float3 D = ray_dir * l;
    float radius = sphere->radius;

    float b = dot(O, D);
    float c = dot(O, O)-radius*radius;
    float disc = b*b-c;
    if(disc > 0.0f)
    {
        float sdisc = sqrtf(disc);
        float root1 = (-b - sdisc);

        bool do_refine = false;

        float root11 = 0.0f;

        if(use_robust_method && fabsf(root1) > 10.f * radius)
        {
             do_refine = true;
        }

        if(do_refine) {
            // refine root1
            float3 O1 = O + root1 * D;
            b = dot(O1, D);
            c = dot(O1, O1) - radius*radius;
            disc = b*b - c;

            if(disc > 0.0f)
            {
                sdisc = sqrtf(disc);
                root11 = (-b - sdisc);
            }
        }

        bool check_second = true;

        float  t;
        float3 normal;
        t = (root1 + root11) * l;
        if ( t > ray_tmin && t < ray_tmax )
        {
            normal = (O + (root1 + root11)*D)/radius;
            if (optixReportIntersection( t, 0, float3_as_args( normal ) ) )
                check_second = false;
        }

        if(check_second)
        {
            float root2 = (-b + sdisc) + (do_refine ? root1 : 0);
            t = root2 * l;
            normal = (O + root2*D)/radius;
            if ( t > ray_tmin && t < ray_tmax )
                optixReportIntersection( t, 0, float3_as_args( normal ) );
        }
    }
}

extern "C" __global__ void __intersection__sphere()
{
    intersect_sphere();
}

extern "C" __global__ void __intersection__sphere_shell()
{
    const SphereShell* sphere_shell = reinterpret_cast<SphereShell*>( optixGetSbtDataPointer() );
    const float3  ray_orig = optixGetWorldRayOrigin();
    const float3  ray_dir  = optixGetWorldRayDirection();
    const float   ray_tmin = optixGetRayTmin(), ray_tmax = optixGetRayTmax();

    float3 O = ray_orig - sphere_shell->center;
    float  l = 1 / length(ray_dir);
    float3 D = ray_dir * l;

    float b = dot(O, D), sqr_b = b * b;
    float O_dot_O = dot(O, O);
    float radius1 = sphere_shell->radius1, radius2 = sphere_shell->radius2;
    float sqr_radius1 = radius1 * radius1, sqr_radius2 = radius2*radius2;

    // check if we are outside of outer sphere
    if ( O_dot_O > sqr_radius2 + params.scene_epsilon )
    {
        if ( O_dot_O - sqr_b < sqr_radius2 - params.scene_epsilon )
        {
            float c = O_dot_O - sqr_radius2;
            float root = sqr_b - c;
            if (root > 0.0f) {
                float t = -b - sqrtf( root );
                float3 normal = (O + t * D) / radius2;
                optixReportIntersection(
                    t * l,
                    HIT_OUTSIDE_FROM_OUTSIDE,
                    float3_as_args( normal ) );
            }
        }
    }
    // else we are inside of the outer sphere
    else
    {
        float c = O_dot_O - sqr_radius1;
        float root = b*b-c;
        if ( root > 0.0f )
        {
            float t = -b - sqrtf( root );
            // do we hit inner sphere from between spheres?
            if ( t * l > ray_tmin && t * l < ray_tmax )
            {
                float3 normal = (O + t * D) / (-radius1);
                optixReportIntersection(
                    t * l,
                    HIT_INSIDE_FROM_OUTSIDE,
                    float3_as_args( normal ) );
            }
            else
            {
                // do we hit inner sphere from within both spheres?
                t = -b + (root > 0 ? sqrtf( root ) : 0.f);
                if ( t * l > ray_tmin && t * l < ray_tmax )
                {
                    float3 normal = ( O + t*D )/(-radius1);
                    optixReportIntersection(
                        t * l,
                        HIT_INSIDE_FROM_INSIDE,
                        float3_as_args( normal ) );
                }
                else
                {
                    // do we hit outer sphere from between spheres?
                    c = O_dot_O - sqr_radius2;
                    root = b*b-c;
                    t = -b + (root > 0 ? sqrtf( root ) : 0.f);
                    float3 normal = ( O + t*D )/radius2;
                    optixReportIntersection(
                        t * l,
                        HIT_OUTSIDE_FROM_INSIDE,
                        float3_as_args( normal ) );
                }
            }
        }
        else
        {
            // do we hit outer sphere from between spheres?
            c = O_dot_O - sqr_radius2;
            root = b*b-c;
            float t = -b + (root > 0 ? sqrtf( root ) : 0.f);
            float3 normal = ( O + t*D )/radius2;
            optixReportIntersection(
                t * l,
                HIT_OUTSIDE_FROM_INSIDE,
                float3_as_args( normal ) );
        }
    }
}
