#include "hip/hip_runtime.h"
#include <optix.h>
#include "OptixPreprocessor.h"
#include "OptixMath.h"
#include "OptixBufferView.h"
#include "OptixLight.h"
#include "OptixGeometry.h"
#include "OptixMaterialData.h"
#include "OptixRecordData.h"
#include "OptixLaunchParams.h"
#include <string>

extern "C" {
__constant__ PickParams params;
}

static __forceinline__ __device__ void pick(
        OptixTraversableHandle      handle,
        float3                      ray_origin,
        float3                      ray_direction,
        float                       tmin,
        float                       tmax,
        PickingPayload*   payload
        )
{
    uint32_t u0=0, u1=0;
    optixTrace(
            handle,
            ray_origin, ray_direction,
            tmin,
            tmax,
            0.0f, // rayTime
            OptixVisibilityMask( 1 ),
            OPTIX_RAY_FLAG_NONE,
            PICK_RAY_TYPE_PICK,     // SBT offset
            PICK_RAY_TYPE_COUNT,    // SBT stride
            PICK_RAY_TYPE_PICK,     // missSBTIndex
            u0, u1);

	payload->meshID = u0;
	payload->primitiveID = u1;
}

extern "C" __global__ void __raygen__pickRay()
{
	if(!params.picking)	{return;}

	// trace pick ray
    PickingPayload payload;
	payload.meshID = ~0;
	payload.primitiveID = ~0;
   
	pick(
            params.sceneAccel,
            params.rayOrigin,
            params.rayDir,
            0.01f,  // tmin       // TODO: smarter offset
            1e16f,  // tmax
            &payload );
			
	unsigned int* pickData = reinterpret_cast<unsigned int*>(params.pickBuffer);

	pickData[0] = payload.meshID;
	pickData[1] = payload.primitiveID;
  
	//printf("Picked mesh: %i\n", pickData[0] ); 
	// printf("Picked triangle:  %i\n", pickData[1]); 
}

extern "C" __global__ void __closesthit__pickHit()
{
	 unsigned int primID = optixGetPrimitiveIndex();
	 unsigned int instID = optixGetInstanceId();
	 
	 optixSetPayload_0( static_cast<unsigned int>( instID ) );
	 optixSetPayload_1( static_cast<unsigned int>( primID ) );
}

extern "C" __global__ void __miss__pickMiss()
{
	optixSetPayload_0( static_cast<unsigned int>( ~0 ) );
	optixSetPayload_1( static_cast<unsigned int>( ~0 ) );
   
}
	